#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

using namespace std;

 
struct lex_comparator
{
  __host__ __device__
  bool operator()(int x, int y)
  {
    unsigned numOfDigits;
    
    if (x > y) {
     numOfDigits = log10f(x) + 1;
    }
    else
    {
     numOfDigits = log10f(y) + 1;
    }
 
 
    for(int i=0; i < numOfDigits; i++, x/=10, y/10)
    { 
      if(x % 10 == y % 10)
      {
        // do nothing
      }
      else if(x % 10 > y % 10)
      {
        return false;
      }
	  else
	  {
        return true;
      }                    
    }

    return false;
  }
};
 
 
/*
Sorts the file, line by line lexicographically.
*/
int main() {

  ifstream ifile("col1.txt");
  int num = 3;
  
  // create host vector of size num
  thrust::host_vector<int> H(num);
  
  // read file
  istream_iterator<int> beg(ifile), end;
 
  // copy into host vecotr
  thrust::copy(beg, end, H.begin());
 
  // crete device vector
  thrust::device_vector<int> D(num);
 
  // copy host vector to device vector
  thrust::copy(H.begin(), H.end(), D.begin());
 
  ifile.close();

  // thrust::sort(D.begin(), D.end());
  thrust::sort(D.begin(), D.end(), lex_comparator());
 
  // copy device vector to host vector
  thrust::copy(D.begin(), D.end(), H.begin());
 
 
  // create output file
  ofstream ofile("sort.txt");
 
  // copy host vector to output file
  thrust::copy(H.begin(), H.end(), ostream_iterator<int>(ofile, "\n"));

  ofile.close();
 
  return 0;
}