#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

using namespace std;

/*
Compares two integers lexicographically from least to greatest. 

First the input integers are reversed. Next reversed integers are traversed from right to left using mod 10 and divide 10 operations. The operator returns as soon as a digit is found that is not the same.

If all digits compared in the loop are the same, then the lengths of the input integers are compared and the operator returns the longer of the two input integers as being greater.
*/
struct lex_comparator
{
  __host__ __device__
  bool operator()(int x, int y)
  {
    unsigned numOfDigits;
  
    if (x > y) {
     numOfDigits = log10f(x) + 1;
    }
    else
    {
     numOfDigits = log10f(y) + 1;
    }
 
    int rX = 0, rY = 0, remainderX, remainderY;
 
    while(x != 0)
    {
        remainderX = x%10;
        rX = rX*10 + remainderX;
        x /= 10;
    }
 
     while(y != 0)
    {
        remainderY = y%10;
        rY = rY*10 + remainderY;
        y /= 10;
    }
 

    for(int i=0; i < numOfDigits; i++, rX/=10, rY/=10)
    { 
      if(rX % 10 == rY % 10)
      {
      // do nothing
      }
      else if(rX % 10 > rY % 10)
      {
        return false;
      }
      else
      {
        return true;
      }  
    }
 
 
   int numOfDigitsX = log10f(x) + 1;
   int numOfDigitsY = log10f(y) + 1;
 
 
   if(numOfDigitsY < numOfDigitsX) {
      return true;
   } else {
     return false;
   }
 
  }
};
 
 
/*
Sorts the file, line by line lexicographically.
*/
int main(int argc, char* argv[]) 
{

  cout << "Running " << argv[0] << '\n';
  cout << "Input file: " << argv[1] << '\n';
  cout << "Number of values: " << argv[2] << '\n';
 
  ifstream ifile(argv[1]);
  int num = std::atoi(argv[2]);
  
  // create host vector of size num
  thrust::host_vector<int> H(num);
  
  // read file
  istream_iterator<int> beg(ifile), end;
 
  // copy into host vecotr
  thrust::copy(beg, end, H.begin());
 
  // crete device vector
  thrust::device_vector<int> D(num);
 
  // copy host vector to device vector
  thrust::copy(H.begin(), H.end(), D.begin());
 
  ifile.close();

  // thrust::sort(D.begin(), D.end());
  thrust::sort(D.begin(), D.end(), lex_comparator());
 
  // copy device vector to host vector
  thrust::copy(D.begin(), D.end(), H.begin());
 
 
  // create output file
  ofstream ofile("sort.txt");
 
  // copy host vector to output file
  thrust::copy(H.begin(), H.end(), ostream_iterator<int>(ofile, "\n"));

  ofile.close();
 
  return 0;
}