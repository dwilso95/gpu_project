#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/random.h>

using namespace std;

/*
Compares two integers lexicographically from least to greatest. 

First the input integers are reversed. Next reversed integers are traversed from right to left using mod 10 and divide 10 operations. The operator returns as soon as a digit is found that is not the same.

If all digits compared in the loop are the same, then the lengths of the input integers are compared and the operator returns the longer of the two input integers as being greater.
*/
struct lex_comparator
{
  __host__ __device__
  bool operator()(thrust::pair<int, char*> xPair, thrust::pair<int, char*> yPair)
  {
    int x = xPair.first;
    int y = yPair.first;
    unsigned numOfDigits;
  
    if (x > y) {
     numOfDigits = log10f(x) + 1;
    }
    else
    {
     numOfDigits = log10f(y) + 1;
    }
 
    int rX = 0, rY = 0, remainderX, remainderY;
 
    while(x != 0)
    {
        remainderX = x%10;
        rX = rX*10 + remainderX;
        x /= 10;
    }
 
     while(y != 0)
    {
        remainderY = y%10;
        rY = rY*10 + remainderY;
        y /= 10;
    }
 

    for(int i=0; i < numOfDigits; i++, rX/=10, rY/=10)
    { 
      if(rX % 10 == rY % 10)
      {
      // do nothing
      }
      else if(rX % 10 > rY % 10)
      {
        return false;
      }
      else
      {
        return true;
      }  
    }
 
 
   int numOfDigitsX = log10f(x) + 1;
   int numOfDigitsY = log10f(y) + 1;
 
 
   if(numOfDigitsY < numOfDigitsX) {
      return true;
   } else {
     return false;
   }
 
  }
};
 
                                  
void print(const thrust::device_vector< thrust::pair<int, char*> >& v)
{
  for(size_t i = 0; i < v.size(); i++)
  {
    thrust::pair<int, char*> p = v[i];
    cout << "Column value: " << p.first << " => Row: " << p.second << "\n";
  }
  cout << "\n";
}
 
void printToFile(const thrust::device_vector< thrust::pair<int, char*> >& v, char* file)
{
  ofstream myfile;
  myfile.open (file);
 
  for(size_t i = 0; i < v.size(); i++)
  {
    thrust::pair<int, char*> p = v[i];
    myfile << p.first << " " << p.second << '\n';
  }
  myfile.close();
}


                                  
void initializePairs(thrust::device_vector< thrust::pair<int, char*> >& v, char* inputFile, char* colfile)
{
  // read file
  ifstream ifile(colfile);
  istream_iterator<int> beg(ifile); 
  std::ifstream infile(inputFile);

  std::string line;
  int i = 0;
  while (std::getline(infile, line))
  {
    char *cstr = new char[line.length() + 1];
    strcpy(cstr, line.c_str());
    v[i++] = thrust::make_pair(*beg++, cstr);
  }
}
 
 
/*
Sorts the input file, line by line lexicographically based on the given column file.
 
Results in a file of pairs. Each pair has a key of column value and value of entire row.
*/
int main(int argc, char* argv[]) 
{

  cout << "Running " << argv[0] << '\n';
  cout << "Input file: " << argv[1] << '\n';
  cout << "Column file: " << argv[2] << '\n';
  cout << "Number of values: " << argv[3] << '\n';
  cout << "Output file: " << argv[4] << '\n';
 
  // num of columns
  int num = std::atoi(argv[3]);
  
  // device vector
  thrust::device_vector< thrust::pair<int, char*> > pairs(num);
  
  // initialie device vector from files
  initializePairs(pairs, argv[1], argv[2]);
  
  // for debugging
  print(pairs);
  
  // sort by key using lex comparator
  thrust::sort(pairs.begin(), pairs.end(), lex_comparator());
  
  // print output to file
  printToFile(pairs, argv[4]);

  return 0;
}
